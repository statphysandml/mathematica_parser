#include "hip/hip_runtime.h"
#include "three_point_system_jacobian_equation.hpp"

std::string ThreePointSystemJacobianEquations::model_ = "three_point_system";
size_t ThreePointSystemJacobianEquations::dim_ = 3;


struct comp_func_three_point_system3
{
	const cudaT const_expr0_;

	comp_func_three_point_system3(const cudaT const_expr0)
		: const_expr0_(const_expr0) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::get<3>(t) = -2 + (thrust::get<2>(t) * (pow((1 + thrust::get<0>(t)), -4)) * (1 + (8 * thrust::get<0>(t)) + (-16 * thrust::get<1>(t) * (-1 + (4 * thrust::get<1>(t)) + thrust::get<0>(t)))) * const_expr0_);
	}
};


void ThreePointSystemJacobianEquation0::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(variables[0].begin(), variables[1].begin(), variables[2].begin(), derivatives.begin())),thrust::make_zip_iterator(thrust::make_tuple(variables[0].end(), variables[1].end(), variables[2].end(), derivatives.end())), comp_func_three_point_system3(const_expr0_));
}


struct comp_func_three_point_system4
{
	const cudaT const_expr0_;

	comp_func_three_point_system4(const cudaT const_expr0)
		: const_expr0_(const_expr0) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::get<3>(t) = thrust::get<2>(t) * (pow((1 + thrust::get<0>(t)), -3)) * (-1 + (16 * thrust::get<1>(t)) + (3 * thrust::get<0>(t))) * const_expr0_;
	}
};


void ThreePointSystemJacobianEquation1::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(variables[0].begin(), variables[1].begin(), variables[2].begin(), derivatives.begin())),thrust::make_zip_iterator(thrust::make_tuple(variables[0].end(), variables[1].end(), variables[2].end(), derivatives.end())), comp_func_three_point_system4(const_expr0_));
}


struct comp_func_three_point_system5
{
	const cudaT const_expr0_;

	comp_func_three_point_system5(const cudaT const_expr0)
		: const_expr0_(const_expr0) {}

	__host__ __device__
	cudaT operator()(const cudaT &val1, const cudaT &val2)
	{
		return (pow((1 + val1), -3)) * (-17 + (64 * (pow(val2, 2))) + (8 * val2 * (-1 + (3 * val1))) + (-12 * val1 * (4 + (val1 * (3 + val1))))) * const_expr0_;
	}
};


void ThreePointSystemJacobianEquation2::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::transform(variables[0].begin(), variables[0].end(), variables[1].begin(), derivatives.begin(), comp_func_three_point_system5(const_expr0_));
}


struct comp_func_three_point_system6
{
	const cudaT const_expr0_;

	comp_func_three_point_system6(const cudaT const_expr0)
		: const_expr0_(const_expr0) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::get<3>(t) = thrust::get<2>(t) * (pow((1 + thrust::get<0>(t)), -6)) * ((160 * (pow(thrust::get<1>(t), 4)) * (289 + (216 * thrust::get<0>(t)))) + (-57 * (1 + thrust::get<0>(t)) * (-1 + (10 * thrust::get<0>(t) * (2 + thrust::get<0>(t))))) + (-40 * (pow(thrust::get<1>(t), 3)) * (424 + (thrust::get<0>(t) * (14 + (45 * thrust::get<0>(t)))))) + (20 * (pow(thrust::get<1>(t), 2)) * (1220 + (thrust::get<0>(t) * (1546 + (771 * thrust::get<0>(t)))))) + (5 * thrust::get<1>(t) * (-1278 + (thrust::get<0>(t) * (-1171 + (3 * thrust::get<0>(t) * (3 + (67 * thrust::get<0>(t))))))))) * const_expr0_;
	}
};


void ThreePointSystemJacobianEquation3::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(variables[0].begin(), variables[1].begin(), variables[2].begin(), derivatives.begin())),thrust::make_zip_iterator(thrust::make_tuple(variables[0].end(), variables[1].end(), variables[2].end(), derivatives.end())), comp_func_three_point_system6(const_expr0_));
}


struct comp_func_three_point_system7
{
	const cudaT const_expr0_;
	const cudaT const_expr1_;

	comp_func_three_point_system7(const cudaT const_expr0, const cudaT const_expr1)
		: const_expr0_(const_expr0), const_expr1_(const_expr1) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::get<3>(t) = (pow((1 + thrust::get<0>(t)), -5)) * (((pow((1 + thrust::get<0>(t)), 5)) * const_expr0_) + (thrust::get<2>(t) * (3113 + (-256 * (pow(thrust::get<1>(t), 3)) * (343 + (270 * thrust::get<0>(t)))) + (240 * (pow(thrust::get<1>(t), 2)) * (87 + (thrust::get<0>(t) * (11 + (15 * thrust::get<0>(t)))))) + (-80 * thrust::get<1>(t) * (347 + (thrust::get<0>(t) * (515 + (257 * thrust::get<0>(t)))))) + (5 * thrust::get<0>(t) * (557 + (3 * thrust::get<0>(t) * (-19 + (thrust::get<0>(t) * (-17 + (5 * thrust::get<0>(t) * (5 + thrust::get<0>(t)))))))))))) * const_expr1_;
	}
};


void ThreePointSystemJacobianEquation4::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(variables[0].begin(), variables[1].begin(), variables[2].begin(), derivatives.begin())),thrust::make_zip_iterator(thrust::make_tuple(variables[0].end(), variables[1].end(), variables[2].end(), derivatives.end())), comp_func_three_point_system7(const_expr0_, const_expr1_));
}


struct comp_func_three_point_system8
{
	const cudaT const_expr0_;

	comp_func_three_point_system8(const cudaT const_expr0)
		: const_expr0_(const_expr0) {}

	__host__ __device__
	cudaT operator()(const cudaT &val1, const cudaT &val2)
	{
		return (pow((1 + val1), -5)) * ((-128 * (pow(val2, 4)) * (343 + (270 * val1))) + (160 * (pow(val2, 3)) * (87 + (val1 * (11 + (15 * val1))))) + (-80 * (pow(val2, 2)) * (347 + (val1 * (515 + (257 * val1))))) + (57 * (1 + val1) * (33 + (4 * val1 * (2 + val1) * (17 + (6 * val1 * (2 + val1)))))) + (2 * val2 * (3113 + (5 * val1 * (557 + (3 * val1 * (-19 + (val1 * (-17 + (5 * val1 * (5 + val1))))))))))) * const_expr0_;
	}
};


void ThreePointSystemJacobianEquation5::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::transform(variables[0].begin(), variables[0].end(), variables[1].begin(), derivatives.begin(), comp_func_three_point_system8(const_expr0_));
}


struct comp_func_three_point_system9
{
	const cudaT const_expr0_;

	comp_func_three_point_system9(const cudaT const_expr0)
		: const_expr0_(const_expr0) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::get<3>(t) = (pow(thrust::get<2>(t), 2)) * (pow((1 + thrust::get<0>(t)), -6)) * (252 + (-32 * (pow(thrust::get<1>(t), 3)) * (289 + (216 * thrust::get<0>(t)))) + (8 * (pow(thrust::get<1>(t), 2)) * (367 + (thrust::get<0>(t) * (-43 + (45 * thrust::get<0>(t)))))) + (thrust::get<0>(t) * (145 + (3 * thrust::get<0>(t) * (111 + (47 * thrust::get<0>(t)))))) + (-4 * thrust::get<1>(t) * (308 + (thrust::get<0>(t) * (-50 + (87 * thrust::get<0>(t))))))) * const_expr0_;
	}
};


void ThreePointSystemJacobianEquation6::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(variables[0].begin(), variables[1].begin(), variables[2].begin(), derivatives.begin())),thrust::make_zip_iterator(thrust::make_tuple(variables[0].end(), variables[1].end(), variables[2].end(), derivatives.end())), comp_func_three_point_system9(const_expr0_));
}


struct comp_func_three_point_system10
{
	const cudaT const_expr0_;

	comp_func_three_point_system10(const cudaT const_expr0)
		: const_expr0_(const_expr0) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::get<3>(t) = (pow(thrust::get<2>(t), 2)) * (pow((1 + thrust::get<0>(t)), -5)) * ((24 * (pow(thrust::get<1>(t), 2)) * (343 + (270 * thrust::get<0>(t)))) + (5 * (62 + (thrust::get<0>(t) * (2 + (29 * thrust::get<0>(t)))))) + (-5 * thrust::get<1>(t) * (291 + (thrust::get<0>(t) * (-13 + (60 * thrust::get<0>(t))))))) * const_expr0_;
	}
};


void ThreePointSystemJacobianEquation7::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(variables[0].begin(), variables[1].begin(), variables[2].begin(), derivatives.begin())),thrust::make_zip_iterator(thrust::make_tuple(variables[0].end(), variables[1].end(), variables[2].end(), derivatives.end())), comp_func_three_point_system10(const_expr0_));
}


struct comp_func_three_point_system11
{
	const cudaT const_expr0_;
	const cudaT const_expr1_;

	comp_func_three_point_system11(const cudaT const_expr0, const cudaT const_expr1)
		: const_expr0_(const_expr0), const_expr1_(const_expr1) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::get<3>(t) = (pow((1 + thrust::get<0>(t)), -5)) * (((pow((1 + thrust::get<0>(t)), 5)) * const_expr0_) + (2 * thrust::get<2>(t) * (-833 + (64 * (pow(thrust::get<1>(t), 3)) * (343 + (270 * thrust::get<0>(t)))) + (40 * thrust::get<1>(t) * (62 + (thrust::get<0>(t) * (2 + (29 * thrust::get<0>(t)))))) + (-20 * (pow(thrust::get<1>(t), 2)) * (291 + (thrust::get<0>(t) * (-13 + (60 * thrust::get<0>(t)))))) + (-5 * thrust::get<0>(t) * (329 + (3 * thrust::get<0>(t) * (171 + (thrust::get<0>(t) * (97 + (5 * thrust::get<0>(t) * (5 + thrust::get<0>(t)))))))))))) * const_expr1_;
	}
};


void ThreePointSystemJacobianEquation8::operator() (odesolver::DimensionIteratorC &derivatives, const odesolver::DevDatC &variables)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(variables[0].begin(), variables[1].begin(), variables[2].begin(), derivatives.begin())),thrust::make_zip_iterator(thrust::make_tuple(variables[0].end(), variables[1].end(), variables[2].end(), derivatives.end())), comp_func_three_point_system11(const_expr0_, const_expr1_));
}

